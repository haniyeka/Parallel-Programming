#include "hip/hip_runtime.h"
﻿#include "DataLayout.h"
#include "GPUErrors.h"

__host__ void GPUAoS(AoS* h_in, AoS* h_out, int size)
{
	AoS* d_in, * d_out;
	const int SizeInBytes_AoS = LEN * sizeof(AoS);
	cout << "\n\nGPU Execution using Array of Structures (AoS)" << endl;
	cout << "\tArray of Structures (AoS) Size in Bytes = " << SizeInBytes_AoS << " bytes" << endl;

	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	float fElapsedTime;

	HandleCUDAError(hipEventCreate(&kernel_start));
	HandleCUDAError(hipEventCreate(&kernel_stop));

	//Allocate device memory on the global memory
	HandleCUDAError(hipMalloc((AoS**)&d_in, SizeInBytes_AoS));
	HandleCUDAError(hipMalloc((AoS**)&d_out, SizeInBytes_AoS));

	//Transfer data from CPU Memory to GPU Memory
	HandleCUDAError(hipMemcpy(d_in, h_in, SizeInBytes_AoS, hipMemcpyHostToDevice));

	//Kernel Invoke Parameters - 1D Grid and 1D Blocks
	int dimx = 256;
	dim3 block(dimx, 1);
	dim3 grid((size + block.x - 1) / block.x, 1);

	cout << "\t1D Grid Dimension" << endl;
	cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
	cout << "\t1D Block Dimension" << endl;
	cout << "\tNumber of threads along X dimension: " << block.x << endl;

	HandleCUDAError(hipEventRecord(kernel_start));
	gpuComputeAoS << <grid, block >> > (d_in, d_out, size);
	HandleCUDAError(hipEventRecord(kernel_stop));
	HandleCUDAError(hipEventSynchronize(kernel_stop));
	GetCUDARunTimeError();
	HandleCUDAError(hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop));

	HandleCUDAError(hipMemcpy(h_out, d_out, SizeInBytes_AoS, hipMemcpyDeviceToHost));
	cout << "\tGPU Computation using Array of Structures (AoS): Elapsed Time = " << fElapsedTime << " msecs" << endl;

	HandleCUDAError(hipFree(d_in));
	HandleCUDAError(hipFree(d_out));

	HandleCUDAError(hipEventDestroy(kernel_start));
	HandleCUDAError(hipEventDestroy(kernel_stop));
	HandleCUDAError(hipDeviceReset());
}

__global__ void gpuComputeAoS(AoS* g_in, AoS* g_out, int n)
{
	unsigned int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	if (idx < n)
	{
		AoS temp = g_in[idx];
		temp.x += 10.0f;
		temp.y += 20.0f;
		g_out[idx] = temp;
	}
}