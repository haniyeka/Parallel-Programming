#include "hip/hip_runtime.h"
﻿#include "DataLayout.h"
#include "GPUErrors.h"

__host__ void GPUSoA(SoA* h_in, SoA* h_out, int size)
{
	SoA* d_in, * d_out;
	const int SizeInBytes_SoA = sizeof(SoA);
	cout << "\n\nGPU Execution using Structure of Arrays (SoA)" << endl;
	cout << "\tStructure of Arrays (SoA) Size in Bytes = " << SizeInBytes_SoA << " bytes" << endl;

	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	float fElapsedTime;

	HandleCUDAError(hipEventCreate(&kernel_start));
	HandleCUDAError(hipEventCreate(&kernel_stop));

	//Allocate device memory on the global memory
	HandleCUDAError(hipMalloc((SoA**)&d_in, SizeInBytes_SoA));
	HandleCUDAError(hipMalloc((AoS**)&d_out, SizeInBytes_SoA));

	//Transfer data from CPU Memory to GPU Memory
	HandleCUDAError(hipMemcpy(d_in, h_in, SizeInBytes_SoA, hipMemcpyHostToDevice));

	//Kernel Invoke Parameters - 1D Grid and 1D Blocks
	int dimx = 256;
	dim3 block(dimx, 1);
	dim3 grid((size + block.x - 1) / block.x, 1);

	cout << "\t1D Grid Dimension" << endl;
	cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
	cout << "\t1D Block Dimension" << endl;
	cout << "\tNumber of threads along X dimension: " << block.x << endl;

	HandleCUDAError(hipEventRecord(kernel_start));
	gpuComputeSoA << <grid, block >> > (d_in, d_out, size);
	HandleCUDAError(hipEventRecord(kernel_stop));
	HandleCUDAError(hipEventSynchronize(kernel_stop));
	GetCUDARunTimeError();
	HandleCUDAError(hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop));

	HandleCUDAError(hipMemcpy(h_out, d_out, SizeInBytes_SoA, hipMemcpyDeviceToHost));
	cout << "\tGPU Computation using Structure of Arrays (SoA): Elapsed Time = " << fElapsedTime << " msecs" << endl;

	HandleCUDAError(hipFree(d_in));
	HandleCUDAError(hipFree(d_out));

	HandleCUDAError(hipEventDestroy(kernel_start));
	HandleCUDAError(hipEventDestroy(kernel_stop));
	HandleCUDAError(hipDeviceReset());
}

__global__ void gpuComputeSoA(SoA* g_in, SoA* g_out, int n)
{
	unsigned int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	if (idx < n)
	{
		float tempX = g_in->x[idx];
		float tempY = g_in->y[idx];

		tempX += 10.0f;
		tempY += 20.0f;

		g_out->x[idx] = tempX;
		g_out->y[idx] = tempY;
	}
}